//#include <limits.h>
//#include <stdint.h>
//#include <asm-generic/errno.h>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
//#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
//#include <stdbool.h>

const long long int DEFAULT_ARRAY_SIZE = 100000000;
const int DEFAULT_RUNS = 2;
const int DEFAULT_THREADS = 256;
const int DEFAULT_BLOCKS = 8;

// Код взят из
// https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
// Очень интересная и полезная презентация
template <unsigned int blockSize>
__device__ void warpReduce(volatile int *sdata, int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__global__ void reduce6(int *g_idata, int *g_odata, const long long int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned long long int i = blockIdx.x*(blockSize*2) + tid;
    unsigned int gridSize = blockSize*2*gridDim.x;
    sdata[tid] = 0;
    while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+blockSize]; i += gridSize; }
    __syncthreads();
    if (blockSize >= 1024) { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int* CreateArray( const long long int SIZE) {
    int* llint_array = (int*) malloc(sizeof(int) * SIZE);
    for (int i = 0; i < SIZE; i++) {
        llint_array[i] = rand()%100;
    }
    return llint_array;
}

long long int GetEnvArraySize() {
    char* array_size_char = getenv("ARRAY_SIZE");
    long long int array_size_int = DEFAULT_ARRAY_SIZE;
    if (array_size_char != NULL) {
        array_size_int = atoll(array_size_char);
    } else {
        printf(
            "Переменная среды ARRAY_SIZE не получена, "
            "используем значение по умолчанию: %lld \n", DEFAULT_ARRAY_SIZE
        );
    }
    return array_size_int;
}

int GetEnvThreads() {
    char* thread_char = getenv("THREADS");
    int thread_int = DEFAULT_THREADS;
    if (thread_char != NULL) {
        thread_int = atoi(thread_char);
    } else {
        printf(
            "Переменная среды THREADS не получена, "
            "используем значение по умолчанию: %d \n", DEFAULT_THREADS
        );
    }
    return thread_int;
}

// int GetEnvBlocks() {
//     char* block_char = getenv("BLOCKS");
//     int block_int = DEFAULT_BLOCKS;
//     if (block_char != NULL) {
//         block_int = atoi(block_char);
//     } else {
//         printf(
//             "Переменная среды BLOCKS не получена, "
//             "используем значение по умолчанию: %d \n", DEFAULT_BLOCKS
//         );
//     }
//     return block_int;
// }

int GetEnvRuns() {
    char* runs_char = getenv("RUNS");
    int runs_int = DEFAULT_RUNS;
    if (runs_char != NULL) {
        runs_int = atoi(runs_char);
    } else {
        printf(
            "Переменная среды RUNS не получена, "
            "используем значение по умолчанию: %d \n", DEFAULT_RUNS
        );
    }
    return runs_int;
}

void CheckCudaError(hipError_t err){
    if (err != hipSuccess) {
        fprintf(stderr, "Fail (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

long long int SumElementsOfArray(const int* array, const long long int SIZE) {
    long long int result = 0;
    for (long long int i = 0; i < SIZE; i++) {
        result += array[i];
    }
    return result;
}

void PrintArray(const int* array, const long long int SIZE) {
    for (long long int i = 0; i < SIZE; i++) {
        printf("%d ",array[i]);
    }
    printf("\n");
}

int main(int argc, char** argv) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    srand(time(0));
    //srand(1);
    const long long int ARRAY_SIZE = GetEnvArraySize();
    const int RUNS = GetEnvRuns();
    const int THREADS = GetEnvThreads();
    //const int BLOCKS = GetEnvBlocks();
    const int BLOCKS = (ARRAY_SIZE + (2 * THREADS) - 1) / (2 * THREADS);

    printf("\n\nПараллельная программа\n");
    printf("Размер массива: %lld\n", ARRAY_SIZE);
    printf("Выполнений: %d\n", RUNS);
    printf("Потоков в блоке: %d\n", THREADS);
    printf("Блоков (ДЛЯ ДАННОГО ЗАДАНИЯ НАСТРОЙКА КОЛ-ВА БЛОКОВ ИГНОРИРУЕТСЯ,\n\
            ПРОГРАММА САМА ВЫСЧИТАЛА НУЖНОЕ КОЛИЧЕСТВО БЛОКОВ НА ОСНОВЕ КОЛ-ВА ПОТОКОВ): %d\n", BLOCKS);

    int result_array_size = (ARRAY_SIZE + (THREADS-1)) / THREADS;
    
    // Таймер
    struct timespec begin, end;
    double exec_time = 0.0;
    double data_allocation_time = 0.0;

    // Цикл выполнения задачи и подсчёта времени её выполнения
    for (int i = 0; i < RUNS; i++) {

        // Массив хоста с данными
        int* host_float_array = NULL;
        host_float_array = CreateArray(ARRAY_SIZE);

        int* host_result_float_array = NULL;
        host_result_float_array = (int*) malloc(sizeof(int) * result_array_size);

        clock_gettime(CLOCK_REALTIME, &begin); // Начало таймера

        // Выделение глобальной памяти под массив, который будет передан GPU
        int* device_float_array = NULL;
        err = hipMalloc(&device_float_array, ARRAY_SIZE * sizeof(int));
        CheckCudaError(err);
        //printf("Глоб массив выделен\n");

        // Выделение глобальной памяти под массив результат, который будет передан GPU
        int* device_result_float_array = NULL;
        err = hipMalloc(&device_result_float_array, sizeof(int) * result_array_size);
        CheckCudaError(err);
        //printf("Глоб массив результата выделен\n");
        
        //Копирование массива в GPU
        err = hipMemcpy(device_float_array,
                         host_float_array,
                         ARRAY_SIZE * sizeof(int),
                         hipMemcpyHostToDevice
                        );
        CheckCudaError(err);
        //printf("Глоб массив скопирован\n");

        clock_gettime(CLOCK_REALTIME, &end); // Конец таймера
        data_allocation_time += (double)(end.tv_sec - begin.tv_sec) + (double)(end.tv_nsec - begin.tv_nsec)/1e9;
        clock_gettime(CLOCK_REALTIME, &begin); // Начало таймера
        
        // Выполнение задачи
        switch (THREADS) {
            case 1024:
                reduce6<1024><<<BLOCKS, 1024, 1024 * sizeof(int)>>>(device_float_array, device_result_float_array, ARRAY_SIZE);
                break;
            case 512:
                reduce6<512><<<BLOCKS, 512, 512 * sizeof(int)>>>(device_float_array, device_result_float_array, ARRAY_SIZE);
                break;
            case 256:
                reduce6<256><<<BLOCKS, 256, 256 * sizeof(int)>>>(device_float_array, device_result_float_array, ARRAY_SIZE);
                break;
            case 128:
                reduce6<128><<<BLOCKS, 128, 128 * sizeof(int)>>>(device_float_array, device_result_float_array, ARRAY_SIZE);
                break;
        }
        hipDeviceSynchronize();
        err = hipGetLastError();
        CheckCudaError(err);
        //printf("Задача выполнена\n");

        clock_gettime(CLOCK_REALTIME, &end); // Конец таймера
        exec_time += (double)(end.tv_sec - begin.tv_sec) + (double)(end.tv_nsec - begin.tv_nsec)/1e9;
        clock_gettime(CLOCK_REALTIME, &begin); // Начало таймера

        // Берём результат от GPU
        err = hipMemcpy(host_result_float_array,
                         device_result_float_array,
                         result_array_size * sizeof(int),
                         hipMemcpyDeviceToHost
                        );
        CheckCudaError(err);
        //printf("Результат получен\n");
        
        // Освобождаем глобальную память GPU
        err = hipFree(device_float_array);
        CheckCudaError(err);
        err = hipFree(device_result_float_array);
        CheckCudaError(err);
        //printf("Память очищена\n");

        long long int final_device_res = SumElementsOfArray(host_result_float_array, result_array_size);

        clock_gettime(CLOCK_REALTIME, &end); // Конец таймера
        data_allocation_time += (double)(end.tv_sec - begin.tv_sec) + (double)(end.tv_nsec - begin.tv_nsec)/1e9;
        
        //PrintArray(host_result_float_array, result_array_size);
        long long int final_host_res = SumElementsOfArray(host_float_array, ARRAY_SIZE);
        long long int diff = final_host_res - final_device_res;
        printf("Погрешность между вычислением на CPU и GPU (CPU - GPU): %lld\n", diff);
        //printf("сумма на CPU %f\n", final_host_res);
        //printf("сумма на GPU %f\n", final_device_res);

        free(host_float_array);
        free(host_result_float_array);
    }

    double mean_data_alloc_time = data_allocation_time / RUNS;
    double mean_exec_time = exec_time / RUNS;
    printf("Общее время выделения памяти, передачи данных и финального счёта: %f сек. \n", data_allocation_time);
    printf("Среднее время выделения памяти передачи данных и финального счёта: %f сек. \n\n", mean_data_alloc_time);
    printf("Общее время выполнения кода на GPU: %f сек. \n", exec_time);
    printf("Среднее время выполнения кода на GPU: %f сек. \n\n", mean_exec_time );
    printf("Общее время выполнения: %f сек. \n", exec_time + data_allocation_time);
    printf("Среднее время выполнения: %f сек.", mean_exec_time + mean_data_alloc_time);

    return 0;
}
